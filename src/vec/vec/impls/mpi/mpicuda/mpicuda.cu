#include "hip/hip_runtime.h"
#define PETSCVEC_DLL
/*
   This file contains routines for Parallel vector operations.
 */
#include "petscconf.h"
PETSC_CUDA_EXTERN_C_BEGIN
#include "../src/vec/vec/impls/mpi/pvecimpl.h"   /*I  "petscvec.h"   I*/
PETSC_CUDA_EXTERN_C_END
#include "../src/vec/vec/impls/seq/seqcuda/cudavecimpl.h"

#undef __FUNCT__  
#define __FUNCT__ "VecDestroy_MPICUDA"
PetscErrorCode VecDestroy_MPICUDA(Vec v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  delete (CUSPARRAY *)v->spptr;
  ierr = VecDestroy_MPI(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecNorm_MPICUDA"
PetscErrorCode VecNorm_MPICUDA(Vec xin,NormType type,PetscReal *z)
{
  PetscReal      sum,work = 0.0;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (type == NORM_2 || type == NORM_FROBENIUS) {
    ierr = VecNorm_SeqCUDA(xin,NORM_2,&work);
    work *= work;
    ierr = MPI_Allreduce(&work,&sum,1,MPIU_REAL,MPI_SUM,((PetscObject)xin)->comm);CHKERRQ(ierr);
    *z = sqrt(sum);
    ierr = PetscLogFlops(2.0*xin->map->n);CHKERRQ(ierr);
  } else if (type == NORM_1) {
    /* Find the local part */
    ierr = VecNorm_SeqCUDA(xin,NORM_1,&work);CHKERRQ(ierr);
    /* Find the global max */
    ierr = MPI_Allreduce(&work,z,1,MPIU_REAL,MPI_SUM,((PetscObject)xin)->comm);CHKERRQ(ierr);
  } else if (type == NORM_INFINITY) {
    /* Find the local max */
    ierr = VecNorm_SeqCUDA(xin,NORM_INFINITY,&work);CHKERRQ(ierr);
    /* Find the global max */
    ierr = MPI_Allreduce(&work,z,1,MPIU_REAL,MPI_MAX,((PetscObject)xin)->comm);CHKERRQ(ierr);
  } else if (type == NORM_1_AND_2) {
    PetscReal temp[2];
    ierr = VecNorm_SeqCUDA(xin,NORM_1,temp);CHKERRQ(ierr);
    ierr = VecNorm_SeqCUDA(xin,NORM_2,temp+1);CHKERRQ(ierr);
    temp[1] = temp[1]*temp[1];
    ierr = MPI_Allreduce(temp,z,2,MPIU_REAL,MPI_SUM,((PetscObject)xin)->comm);CHKERRQ(ierr);
    z[1] = sqrt(z[1]);
  }
  PetscFunctionReturn(0);
}
#undef __FUNCT__
#define __FUNCT__ "VecView_MPICUDA"
PetscErrorCode VecView_MPICUDA(Vec xin,PetscViewer viewer)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUDACopyFromGPU(xin);CHKERRQ(ierr);
  ierr = VecView_MPI(xin,viewer);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecDot_MPICUDA"
PetscErrorCode VecDot_MPICUDA(Vec xin,Vec yin,PetscScalar *z)
{
  PetscScalar    sum,work;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecDot_SeqCUDA(xin,yin,&work);CHKERRQ(ierr);
  ierr = MPI_Allreduce(&work,&sum,1,MPIU_SCALAR,MPIU_SUM,((PetscObject)xin)->comm);CHKERRQ(ierr);
  *z = sum;
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecTDot_MPICUDA"
PetscErrorCode VecTDot_MPICUDA(Vec xin,Vec yin,PetscScalar *z)
{
  PetscScalar    sum,work;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecTDot_SeqCUDA(xin,yin,&work);CHKERRQ(ierr);
  ierr = MPI_Allreduce(&work,&sum,1,MPIU_SCALAR,MPIU_SUM,((PetscObject)xin)->comm);CHKERRQ(ierr);
  *z   = sum;
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecMDot_MPICUDA"
PetscErrorCode VecMDot_MPICUDA(Vec xin,PetscInt nv,const Vec y[],PetscScalar *z)
{
  PetscScalar    awork[128],*work = awork;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (nv > 128) {
    ierr = PetscMalloc(nv*sizeof(PetscScalar),&work);CHKERRQ(ierr);
  }
  ierr = VecMDot_SeqCUDA(xin,nv,y,work);CHKERRQ(ierr);
  ierr = MPI_Allreduce(work,z,nv,MPIU_SCALAR,MPIU_SUM,((PetscObject)xin)->comm);CHKERRQ(ierr);
  if (nv > 128) {
    ierr = PetscFree(work);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*MC
   VECMPICUDA - VECMPICUDA = "mpicuda" - The basic parallel vector, modified to use CUDA

   Options Database Keys:
. -vec_type mpicuda - sets the vector type to VECMPICUDA during a call to VecSetFromOptions()

  Level: beginner

.seealso: VecCreate(), VecSetType(), VecSetFromOptions(), VecCreateMpiWithArray(), VECMPI, VecType, VecCreateMPI(), VecCreateMpi()
M*/

EXTERN_C_BEGIN
#undef __FUNCT__  
#define __FUNCT__ "VecCreate_MPICUDA"
PetscErrorCode PETSCVEC_DLLEXPORT VecCreate_MPICUDA(Vec vv)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCreate_MPI_Private(vv,PETSC_TRUE,0,0);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)vv,VECMPICUDA);CHKERRQ(ierr);
  vv->valid_GPU_array = PETSC_CUDA_UNALLOCATED;
  vv->ops->dot           = VecDot_MPICUDA;
  vv->ops->mdot          = VecMDot_MPICUDA;
  vv->ops->tdot          = VecTDot_MPICUDA;
  vv->ops->view          = VecView_MPICUDA;
  vv->ops->norm          = VecNorm_MPICUDA;
  vv->ops->scale         = VecScale_SeqCUDA;
  vv->ops->copy          = VecCopy_SeqCUDA;
  vv->ops->set           = VecSet_SeqCUDA;
  vv->ops->swap          = VecSwap_SeqCUDA;
  vv->ops->axpy          = VecAXPY_SeqCUDA;
  vv->ops->axpby         = VecAXPBY_SeqCUDA;
  vv->ops->maxpy         = VecMAXPY_SeqCUDA;
  vv->ops->aypx          = VecAYPX_SeqCUDA;
  vv->ops->axpbypcz      = VecAXPBYPCZ_SeqCUDA;
  vv->ops->pointwisemult = VecPointwiseMult_SeqCUDA;
  vv->ops->setrandom     = VecSetRandom_SeqCUDA;
  vv->ops->replacearray  = VecReplaceArray_SeqCUDA;
  vv->ops->dot_local     = VecDot_SeqCUDA;
  vv->ops->tdot_local    = VecTDot_SeqCUDA;
  vv->ops->norm_local    = VecNorm_SeqCUDA;
  vv->ops->mdot_local    = VecMDot_SeqCUDA;
  vv->ops->destroy       = VecDestroy_MPICUDA;
  /* place array?
     reset array?
     get values?
  */
  PetscFunctionReturn(0);
}
EXTERN_C_END







