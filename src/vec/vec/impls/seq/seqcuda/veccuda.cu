#include "hip/hip_runtime.h"
#define PETSCVEC_DLL
/*
   Implements the sequential vectors.
*/

#include "petscconf.h"
PETSC_CUDA_EXTERN_C_BEGIN
#include "private/vecimpl.h"          /*I "petscvec.h" I*/
#include "../src/vec/vec/impls/dvecimpl.h"
PETSC_CUDA_EXTERN_C_END
#include "../src/vec/vec/impls/seq/seqcuda/cudavecimpl.h"

/* these following 2 public versions are necessary because we use CUSP in the regular version and these need to be called from plain C code. */
#undef __FUNCT__
#define __FUNCT__ "VecCUDAAllocateCheck_Public"
PetscErrorCode VecCUDAAllocateCheck_Public(Vec v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUDAAllocateCheck(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUDACopyToGPU_Public"
PetscErrorCode VecCUDACopyToGPU_Public(Vec v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUDACopyToGPU(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUDACopyFromGPU"
/* Copies a vector from the GPU to the CPU unless we already have an up-to-date copy on the CPU */
PetscErrorCode VecCUDACopyFromGPU(Vec v)
{
  PetscErrorCode ierr;
  CUSPARRAY      *GPUvector;
  PetscScalar    *array;
  Vec_Seq        *s;
  PetscInt       n = v->map->n;

  PetscFunctionBegin;
  s = (Vec_Seq*)v->data;
  if (s->array == 0){
    ierr               = PetscMalloc(n*sizeof(PetscScalar),&array);CHKERRQ(ierr);
    ierr               = PetscLogObjectMemory(v,n*sizeof(PetscScalar));CHKERRQ(ierr);
    s->array           = array;
    s->array_allocated = array;
  }
  if (v->valid_GPU_array == PETSC_CUDA_GPU){
    GPUvector  = ((VecSeqCUDA_Container*)v->spptr)->GPUarray;
    ierr       = PetscLogEventBegin(VEC_CUDACopyFromGPU,v,0,0,0);CHKERRQ(ierr);
    try{
      thrust::copy(GPUvector->begin(),GPUvector->end(),*(PetscScalar**)v->data);
      ierr = WaitForGPU();CHKERRCUDA(ierr);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    ierr = PetscLogEventEnd(VEC_CUDACopyFromGPU,v,0,0,0);CHKERRQ(ierr);
    v->valid_GPU_array = PETSC_CUDA_BOTH;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUDACopyFromGPUSome"
/* Note that this function only copies *some* of the values up from the GPU to CPU,
   which means that we need recombine the data at some point before using any of the standard functions.
   We could add another few flag-types to keep track of this, or treat things like VecGetArray VecRestoreArray
   where you have to always call in pairs 
*/
PetscErrorCode VecCUDACopyFromGPUSome(Vec v)
{
  Vec_Seq        *s;
  PetscInt       n = v->map->n;
  PetscScalar    *array;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  s = (Vec_Seq*)v->data;
  if (s->array == 0){
    ierr               = PetscMalloc(n*sizeof(PetscScalar),&array);CHKERRQ(ierr);
    ierr               = PetscLogObjectMemory(v,n*sizeof(PetscScalar));CHKERRQ(ierr);
    s->array           = array;
    s->array_allocated = array;
  }
  /* now in here we have to do a scatter of some kind */
  PetscFunctionReturn(0);
}




/*MC
   VECSEQCUDA - VECSEQCUDA = "seqcuda" - The basic sequential vector, modified to use CUDA

   Options Database Keys:
. -vec_type seqcuda - sets the vector type to VECSEQCUDA during a call to VecSetFromOptions()

  Level: beginner

.seealso: VecCreate(), VecSetType(), VecSetFromOptions(), VecCreateSeqWithArray(), VECMPI, VecType, VecCreateMPI(), VecCreateSeq()
M*/

/* for VecAYPX_SeqCUDA*/
namespace cusp
{
namespace blas
{
namespace detail
{
  template <typename T>
    struct AYPX : public thrust::binary_function<T,T,T>
    {
      T alpha;
      
      AYPX(T _alpha) : alpha(_alpha) {}

      __host__ __device__
	T operator()(T x, T y)
      {
	return alpha * y + x;
      }
    };
}

 template <typename ForwardIterator1,
           typename ForwardIterator2,
           typename ScalarType>
void aypx(ForwardIterator1 first1,ForwardIterator1 last1,ForwardIterator2 first2,ScalarType alpha)
	   {
	     thrust::transform(first1,last1,first2,first2,detail::AYPX<ScalarType>(alpha));
	   }
 template <typename Array1, typename Array2, typename ScalarType>
   void aypx(const Array1& x, Array2& y, ScalarType alpha)
 {
   detail::assert_same_dimensions(x,y);
   aypx(x.begin(),x.end(),y.begin(),alpha);
 }
}
}

#undef __FUNCT__
#define __FUNCT__ "VecAYPX_SeqCUDA"
PetscErrorCode VecAYPX_SeqCUDA(Vec yin, PetscScalar alpha, Vec xin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (alpha != 0.0) {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
    try{
      cusp::blas::aypx(*((VecSeqCUDA_Container*)xin->spptr)->GPUarray,*((VecSeqCUDA_Container*)yin->spptr)->GPUarray,alpha);
      yin->valid_GPU_array = PETSC_CUDA_GPU;
      ierr = WaitForGPU();CHKERRCUDA(ierr);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    ierr = PetscLogFlops(2.0*yin->map->n);CHKERRQ(ierr);
   }
  PetscFunctionReturn(0);
}

     

#undef __FUNCT__  
#define __FUNCT__ "VecAXPY_SeqCUDA"
PetscErrorCode VecAXPY_SeqCUDA(Vec yin,PetscScalar alpha,Vec xin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  /* assume that the BLAS handles alpha == 1.0 efficiently since we have no fast code for it */
  if (alpha != 0.0) {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
    try {
      cusp::blas::axpy(*((VecSeqCUDA_Container*)xin->spptr)->GPUarray,*((VecSeqCUDA_Container*)yin->spptr)->GPUarray,alpha);
      yin->valid_GPU_array = PETSC_CUDA_GPU;
      ierr = WaitForGPU();CHKERRCUDA(ierr);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    ierr = PetscLogFlops(2.0*yin->map->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

struct VecCUDAPointwiseDivide
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t) / thrust::get<2>(t);
  }
};

#undef __FUNCT__
#define __FUNCT__ "VecPointwiseDivide_SeqCUDA"
PetscErrorCode VecPointwiseDivide_SeqCUDA(Vec win, Vec xin, Vec yin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUDAAllocateCheck(win);CHKERRQ(ierr);
  ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
  ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
  try{
    thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)win->spptr)->GPUarray->begin(),
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->begin(),
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)win->spptr)->GPUarray->end(),  
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->end(),
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->end())),
	VecCUDAPointwiseDivide());
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
  ierr = PetscLogFlops(win->map->n);CHKERRQ(ierr);
  win->valid_GPU_array = PETSC_CUDA_GPU;
  PetscFunctionReturn(0);
}


struct VecCUDAWAXPY
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t) + thrust::get<2>(t)*thrust::get<3>(t);
  }
};

struct VecCUDASum
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t) + thrust::get<2>(t);
  }
};

struct VecCUDADiff
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t) - thrust::get<2>(t);
  }
};

#undef __FUNCT__
#define __FUNCT__ "VecWAXPY_SeqCUDA"
PetscErrorCode VecWAXPY_SeqCUDA(Vec win,PetscScalar alpha,Vec xin, Vec yin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
  ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
  ierr = VecCUDAAllocateCheck(win);CHKERRQ(ierr);
  if (alpha == 1.0) {
    try {
      thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)win->spptr)->GPUarray->begin(),
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->begin(),
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)win->spptr)->GPUarray->end(),  
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->end(),
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->end())),
	VecCUDASum());
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
        ierr = PetscLogFlops(win->map->n);CHKERRQ(ierr);
  } else if (alpha == -1.0) {
    try {
     thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)win->spptr)->GPUarray->begin(),
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->begin(),
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)win->spptr)->GPUarray->end(),  
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->end(),
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->end())),
	VecCUDADiff());
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
        ierr = PetscLogFlops(win->map->n);CHKERRQ(ierr);
  } else if (alpha == 0.0) {
    ierr = VecCopy_SeqCUDA(yin,win);CHKERRQ(ierr);
  } else {
    try {
     thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)win->spptr)->GPUarray->begin(),
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(alpha,0),
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)win->spptr)->GPUarray->end(),  
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->end(),
		thrust::make_constant_iterator(alpha,win->map->n),
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->end())),
	VecCUDAWAXPY());
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
        ierr = PetscLogFlops(2*win->map->n);CHKERRQ(ierr);
  }
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  win->valid_GPU_array = PETSC_CUDA_GPU;
  PetscFunctionReturn(0);
}

/* These functions are for the CUDA implementation of MAXPY with the loop unrolled on the CPU */
struct VecCUDAMAXPY4
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    /*y += a1*x1 +a2*x2 + 13*x3 +a4*x4 */
    thrust::get<0>(t) += thrust::get<1>(t)*thrust::get<2>(t)+thrust::get<3>(t)*thrust::get<4>(t)+thrust::get<5>(t)*thrust::get<6>(t)+thrust::get<7>(t)*thrust::get<8>(t);
  }
};


struct VecCUDAMAXPY3
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    /*y += a1*x1 +a2*x2 + 13*x3 */
    thrust::get<0>(t) += thrust::get<1>(t)*thrust::get<2>(t)+thrust::get<3>(t)*thrust::get<4>(t)+thrust::get<5>(t)*thrust::get<6>(t);
  }
};

struct VecCUDAMAXPY2
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    /*y += a1*x1 +a2*x2*/
    thrust::get<0>(t) += thrust::get<1>(t)*thrust::get<2>(t)+thrust::get<3>(t)*thrust::get<4>(t);
  }
};
#undef __FUNCT__  
#define __FUNCT__ "VecMAXPY_SeqCUDA"
PetscErrorCode VecMAXPY_SeqCUDA(Vec xin, PetscInt nv,const PetscScalar *alpha,Vec *y)
{
  PetscErrorCode    ierr;
  PetscInt          n = xin->map->n,j,j_rem;
  Vec               yy0,yy1,yy2,yy3;
  PetscScalar       alpha0,alpha1,alpha2,alpha3;

  PetscFunctionBegin;
  ierr = PetscLogFlops(nv*2.0*n);CHKERRQ(ierr);
  ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
  switch (j_rem=nv&0x3) {
  case 3: 
    alpha0 = alpha[0]; 
    alpha1 = alpha[1]; 
    alpha2 = alpha[2]; 
    alpha += 3;
    yy0    = y[0];
    yy1    = y[1];
    yy2    = y[2];
    ierr   = VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr   = VecCUDACopyToGPU(yy1);CHKERRQ(ierr);
    ierr   = VecCUDACopyToGPU(yy2);CHKERRQ(ierr);
    try {
      thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(alpha0,0),
		((VecSeqCUDA_Container*)yy0->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(alpha1,0),
		((VecSeqCUDA_Container*)yy1->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(alpha2,0),
		((VecSeqCUDA_Container*)yy2->spptr)->GPUarray->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->end(),  
		thrust::make_constant_iterator(alpha0,n),
		((VecSeqCUDA_Container*)yy0->spptr)->GPUarray->end(),
		thrust::make_constant_iterator(alpha1,n),
		((VecSeqCUDA_Container*)yy1->spptr)->GPUarray->end(),
		thrust::make_constant_iterator(alpha2,n),
		((VecSeqCUDA_Container*)yy2->spptr)->GPUarray->end())),
	VecCUDAMAXPY3());
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    y     += 3;
    break;
  case 2: 
    alpha0 = alpha[0]; 
    alpha1 = alpha[1]; 
    alpha +=2;
    yy0    = y[0];
    yy1    = y[1];
    ierr   = VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr   = VecCUDACopyToGPU(yy1);CHKERRQ(ierr);
    try {
      thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(alpha0,0),
		((VecSeqCUDA_Container*)yy0->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(alpha1,0),
		((VecSeqCUDA_Container*)yy1->spptr)->GPUarray->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->end(),  
		thrust::make_constant_iterator(alpha0,n),
		((VecSeqCUDA_Container*)yy0->spptr)->GPUarray->end(),
		thrust::make_constant_iterator(alpha1,n),
		((VecSeqCUDA_Container*)yy1->spptr)->GPUarray->end())),
	VecCUDAMAXPY2());
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    y     +=2;
    break;
  case 1: 
    alpha0 = *alpha++; 
    yy0 = y[0];
    ierr =  VecAXPY_SeqCUDA(xin,alpha0,yy0);
    y     +=1;
    break;
  }
  for (j=j_rem; j<nv; j+=4) {
    alpha0 = alpha[0];
    alpha1 = alpha[1];
    alpha2 = alpha[2];
    alpha3 = alpha[3];
    alpha  += 4;
    yy0    = y[0];
    yy1    = y[1];
    yy2    = y[2];
    yy3    = y[3];
    ierr   = VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr   = VecCUDACopyToGPU(yy1);CHKERRQ(ierr);
    ierr   = VecCUDACopyToGPU(yy2);CHKERRQ(ierr);
    ierr   = VecCUDACopyToGPU(yy3);CHKERRQ(ierr);
    try {
      thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(alpha0,0),
		((VecSeqCUDA_Container*)yy0->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(alpha1,0),
		((VecSeqCUDA_Container*)yy1->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(alpha2,0),
		((VecSeqCUDA_Container*)yy2->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(alpha3,0),
		((VecSeqCUDA_Container*)yy3->spptr)->GPUarray->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->end(),  
		thrust::make_constant_iterator(alpha0,n),
		((VecSeqCUDA_Container*)yy0->spptr)->GPUarray->end(),
		thrust::make_constant_iterator(alpha1,n),
		((VecSeqCUDA_Container*)yy1->spptr)->GPUarray->end(),
		thrust::make_constant_iterator(alpha2,n),
		((VecSeqCUDA_Container*)yy2->spptr)->GPUarray->end(),
		thrust::make_constant_iterator(alpha3,n),
		((VecSeqCUDA_Container*)yy3->spptr)->GPUarray->end())),
	VecCUDAMAXPY4());
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    y      += 4;
  }
  xin->valid_GPU_array = PETSC_CUDA_GPU;
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  PetscFunctionReturn(0);
} 


#undef __FUNCT__
#define __FUNCT__ "VecDot_SeqCUDA"
PetscErrorCode VecDot_SeqCUDA(Vec xin,Vec yin,PetscScalar *z)
{
#if defined(PETSC_USE_COMPLEX)
  PetscScalar    *ya,*xa;
#endif
  PetscErrorCode ierr;
  PetscFunctionBegin;
#if defined(PETSC_USE_COMPLEX)
  /* cannot use BLAS dot for complex because compiler/linker is 
     not happy about returning a double complex */
  {
    ierr = VecGetArrayPrivate2(xin,&xa,yin,&ya);CHKERRQ(ierr);
    PetscInt    i;
    PetscScalar sum = 0.0;
    for (i=0; i<xin->map->n; i++) {
      sum += xa[i]*PetscConj(ya[i]);
    }
    *z = sum;
    ierr = VecRestoreArrayPrivate2(xin,&xa,yin,&ya);CHKERRQ(ierr);
  }
#else
  {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
    try {
      *z = cusp::blas::dot(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray,*((VecSeqCUDA_Container *)yin->spptr)->GPUarray);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
  }
#endif
  ierr = WaitForGPU();CHKERRCUDA(ierr);
 if (xin->map->n >0) {
    ierr = PetscLogFlops(2.0*xin->map->n-1);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*The following few template functions are for VecMDot_SeqCUDA*/

template <typename T1,typename T2>
struct cudamult2 : thrust::unary_function<T1,T2>
{
	__host__ __device__
	T2 operator()(T1 x)
	{
		return thrust::make_tuple(thrust::get<0>(x)*thrust::get<1>(x),thrust::get<0>(x)*thrust::get<2>(x));
	}
};

template <typename T>
struct cudaadd2 : thrust::binary_function<T,T,T>
{
	__host__ __device__
	T operator()(T x,T y)
	{
		return thrust::make_tuple(thrust::get<0>(x)+thrust::get<0>(y),thrust::get<1>(x)+thrust::get<1>(y));
	}
};
	
template <typename T1,typename T2>
struct cudamult3 : thrust::unary_function<T1,T2>
{
	__host__ __device__
	T2 operator()(T1 x)
	{
	  return thrust::make_tuple(thrust::get<0>(x)*thrust::get<1>(x),thrust::get<0>(x)*thrust::get<2>(x),thrust::get<0>(x)*thrust::get<3>(x));
	}
};

template <typename T>
struct cudaadd3 : thrust::binary_function<T,T,T>
{
	__host__ __device__
	T operator()(T x,T y)
	{
	  return thrust::make_tuple(thrust::get<0>(x)+thrust::get<0>(y),thrust::get<1>(x)+thrust::get<1>(y),thrust::get<2>(x)+thrust::get<2>(y));
	}
};
	template <typename T1,typename T2>
struct cudamult4 : thrust::unary_function<T1,T2>
{
	__host__ __device__
	T2 operator()(T1 x)
	{
	  return thrust::make_tuple(thrust::get<0>(x)*thrust::get<1>(x),thrust::get<0>(x)*thrust::get<2>(x),thrust::get<0>(x)*thrust::get<3>(x),thrust::get<0>(x)*thrust::get<4>(x));
	}
};

template <typename T>
struct cudaadd4 : thrust::binary_function<T,T,T>
{
	__host__ __device__
	T operator()(T x,T y)
	{
	  return thrust::make_tuple(thrust::get<0>(x)+thrust::get<0>(y),thrust::get<1>(x)+thrust::get<1>(y),thrust::get<2>(x)+thrust::get<2>(y),thrust::get<3>(x)+thrust::get<3>(y));
	}
};


#undef __FUNCT__  
#define __FUNCT__ "VecMDot_SeqCUDA"
PetscErrorCode VecMDot_SeqCUDA(Vec xin,PetscInt nv,const Vec yin[],PetscScalar *z)
{
  PetscErrorCode    ierr;
  PetscInt          n = xin->map->n,j,j_rem;
  Vec               yy0,yy1,yy2,yy3;
  PetscScalar       zero=0.0;
  thrust::tuple<PetscScalar,PetscScalar> result2;
  thrust::tuple<PetscScalar,PetscScalar,PetscScalar> result3;
  thrust::tuple<PetscScalar,PetscScalar,PetscScalar,PetscScalar>result4;

  PetscFunctionBegin;
  ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
  switch(j_rem=nv&0x3) {
  case 3: 
    yy0  =  yin[0];
    yy1  =  yin[1];
    yy2  =  yin[2];
    ierr =  VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr =  VecCUDACopyToGPU(yy1);CHKERRQ(ierr);
    ierr =  VecCUDACopyToGPU(yy2);CHKERRQ(ierr);
    try {
      result3 = thrust::transform_reduce(
		     thrust::make_zip_iterator(
			  thrust::make_tuple(
				   ((VecSeqCUDA_Container *)xin->spptr)->GPUarray->begin(),
				   ((VecSeqCUDA_Container *)yy0->spptr)->GPUarray->begin(),
				   ((VecSeqCUDA_Container *)yy1->spptr)->GPUarray->begin(), 
				   ((VecSeqCUDA_Container *)yy2->spptr)->GPUarray->begin())),
		     thrust::make_zip_iterator(
			  thrust::make_tuple(
				   ((VecSeqCUDA_Container *)xin->spptr)->GPUarray->end(),
				   ((VecSeqCUDA_Container *)yy0->spptr)->GPUarray->end(),
				   ((VecSeqCUDA_Container *)yy1->spptr)->GPUarray->end(),
				   ((VecSeqCUDA_Container *)yy2->spptr)->GPUarray->end())),
		     cudamult3<thrust::tuple<PetscScalar,PetscScalar,PetscScalar,PetscScalar>, thrust::tuple<PetscScalar,PetscScalar,PetscScalar> >(),
		     thrust::make_tuple(zero,zero,zero), /*init */
		     cudaadd3<thrust::tuple<PetscScalar,PetscScalar,PetscScalar> >()); /* binary function */
      z[0] = thrust::get<0>(result3);
      z[1] = thrust::get<1>(result3);
      z[2] = thrust::get<2>(result3);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    
    z    += 3;
    yin  += 3;
    break;
  case 2:
    yy0  =  yin[0];
    yy1  =  yin[1];
    ierr =  VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr =  VecCUDACopyToGPU(yy1);CHKERRQ(ierr);
    try {
      result2 = thrust::transform_reduce(
		    thrust::make_zip_iterator(
			thrust::make_tuple(
				  ((VecSeqCUDA_Container *)xin->spptr)->GPUarray->begin(),
				  ((VecSeqCUDA_Container *)yy0->spptr)->GPUarray->begin(),
				  ((VecSeqCUDA_Container *)yy1->spptr)->GPUarray->begin())),
		    thrust::make_zip_iterator(
			thrust::make_tuple(
				  ((VecSeqCUDA_Container *)xin->spptr)->GPUarray->end(),
				  ((VecSeqCUDA_Container *)yy0->spptr)->GPUarray->end(),
				  ((VecSeqCUDA_Container *)yy1->spptr)->GPUarray->end())),
		    cudamult2<thrust::tuple<PetscScalar,PetscScalar,PetscScalar>, thrust::tuple<PetscScalar,PetscScalar> >(),
		    thrust::make_tuple(zero,zero), /*init */
		    cudaadd2<thrust::tuple<PetscScalar, PetscScalar> >()); /* binary function */
      z[0] = thrust::get<0>(result2);
      z[1] = thrust::get<1>(result2);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    z    += 2;
    yin  += 2;
    break;
  case 1: 
    yy0  =  yin[0];
    ierr =  VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr =  VecDot_SeqCUDA(xin,yy0,&z[0]);CHKERRQ(ierr);
    z    += 1;
    yin  += 1;
    break;
  }
  for (j=j_rem; j<nv; j+=4) {
    yy0  =  yin[0];
    yy1  =  yin[1];
    yy2  =  yin[2];
    yy3  =  yin[3];
    ierr =  VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr =  VecCUDACopyToGPU(yy1);CHKERRQ(ierr);
    ierr =  VecCUDACopyToGPU(yy2);CHKERRQ(ierr);
    ierr =  VecCUDACopyToGPU(yy3);CHKERRQ(ierr);
    try {
      result4 = thrust::transform_reduce(
		    thrust::make_zip_iterator(
			thrust::make_tuple(
				  ((VecSeqCUDA_Container *)xin->spptr)->GPUarray->begin(),
				  ((VecSeqCUDA_Container *)yy0->spptr)->GPUarray->begin(),
				  ((VecSeqCUDA_Container *)yy1->spptr)->GPUarray->begin(), 
				  ((VecSeqCUDA_Container *)yy2->spptr)->GPUarray->begin(),
				  ((VecSeqCUDA_Container *)yy3->spptr)->GPUarray->begin())),
		    thrust::make_zip_iterator(
			thrust::make_tuple(
				  ((VecSeqCUDA_Container *)xin->spptr)->GPUarray->end(),
				  ((VecSeqCUDA_Container *)yy0->spptr)->GPUarray->end(),
				  ((VecSeqCUDA_Container *)yy1->spptr)->GPUarray->end(),
				  ((VecSeqCUDA_Container *)yy2->spptr)->GPUarray->end(),
				  ((VecSeqCUDA_Container *)yy3->spptr)->GPUarray->end())),
		     cudamult4<thrust::tuple<PetscScalar,PetscScalar,PetscScalar,PetscScalar,PetscScalar>, thrust::tuple<PetscScalar,PetscScalar,PetscScalar,PetscScalar> >(),
		     thrust::make_tuple(zero,zero,zero,zero), /*init */
		     cudaadd4<thrust::tuple<PetscScalar,PetscScalar,PetscScalar,PetscScalar> >()); /* binary function */
      z[0] = thrust::get<0>(result4);
      z[1] = thrust::get<1>(result4);
      z[2] = thrust::get<2>(result4);
      z[3] = thrust::get<3>(result4);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    z    += 4;
    yin  += 4;
  }  
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = PetscLogFlops(PetscMax(nv*(2.0*n-1),0.0));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecSet_SeqCUDA"
PetscErrorCode VecSet_SeqCUDA(Vec xin,PetscScalar alpha)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  /* if there's a faster way to do the case alpha=0.0 on the GPU we should do that*/
  ierr = VecCUDAAllocateCheck(xin);CHKERRQ(ierr);
  try {
    cusp::blas::fill(*((VecSeqCUDA_Container*)xin->spptr)->GPUarray,alpha);
  } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
  } 
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  xin->valid_GPU_array = PETSC_CUDA_GPU;
  PetscFunctionReturn(0);
}

#undef __FUNCT__ 
#define __FUNCT__ "VecScale_SeqCUDA"
PetscErrorCode VecScale_SeqCUDA(Vec xin, PetscScalar alpha)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (alpha == 0.0) {
    ierr = VecSet_SeqCUDA(xin,alpha);CHKERRQ(ierr);
  } else if (alpha != 1.0) {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    try {
      cusp::blas::scal(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray,alpha);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    xin->valid_GPU_array = PETSC_CUDA_GPU;
  }
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = PetscLogFlops(xin->map->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecTDot_SeqCUDA"
PetscErrorCode VecTDot_SeqCUDA(Vec xin,Vec yin,PetscScalar *z)
{
#if defined(PETSC_USE_COMPLEX)
  PetscScalar    *ya,*xa;
#endif
  PetscErrorCode ierr;

  PetscFunctionBegin;
#if defined(PETSC_USE_COMPLEX)
  /* cannot use BLAS dot for complex because compiler/linker is 
     not happy about returning a double complex */
 ierr = VecGetArrayPrivate2(xin,&xa,yin,&ya);CHKERRQ(ierr);
 {
   PetscInt    i;
   PetscScalar sum = 0.0;
   for (i=0; i<xin->map->n; i++) {
     sum += xa[i]*ya[i];
   }
   *z = sum;
   ierr = VecRestoreArrayPrivate2(xin,&xa,yin,&ya);CHKERRQ(ierr);
 }
#else
 ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
 ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
 try {
   *z = cusp::blas::dot(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray,*((VecSeqCUDA_Container *)yin->spptr)->GPUarray);
 } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
 } 
#endif
 ierr = WaitForGPU();CHKERRCUDA(ierr);
  if (xin->map->n > 0) {
    ierr = PetscLogFlops(2.0*xin->map->n-1);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}
#undef __FUNCT__  
#define __FUNCT__ "VecCopy_SeqCUDA"
PetscErrorCode VecCopy_SeqCUDA(Vec xin,Vec yin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (xin != yin) {
    if (xin->valid_GPU_array == PETSC_CUDA_GPU) {
      /* copy in GPU */
       ierr = VecCUDAAllocateCheck(yin);CHKERRQ(ierr);
       try {
	 cusp::blas::copy(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray,*((VecSeqCUDA_Container *)yin->spptr)->GPUarray);
       } catch(char* ex) {
        SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
      } 
      ierr = WaitForGPU();CHKERRCUDA(ierr);
      yin->valid_GPU_array = PETSC_CUDA_GPU;

    } else if (xin->valid_GPU_array == PETSC_CUDA_CPU || xin->valid_GPU_array == PETSC_CUDA_UNALLOCATED) {
      /* copy in CPU if we are on the CPU*/
      ierr = VecCopy_Seq(xin,yin);CHKERRQ(ierr);
    
    } else if (xin->valid_GPU_array == PETSC_CUDA_BOTH) {
      /* if xin is valid in both places, see where yin is and copy there (because it's probably where we'll want to next use it) */
      if (yin->valid_GPU_array == PETSC_CUDA_CPU) {
	/* copy in CPU */
	ierr = VecCopy_Seq(xin,yin);CHKERRQ(ierr);

      } else if (yin->valid_GPU_array == PETSC_CUDA_GPU) {
	/* copy in GPU */
	ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
	try {
	  cusp::blas::copy(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray,*((VecSeqCUDA_Container *)yin->spptr)->GPUarray);
	  ierr = WaitForGPU();CHKERRCUDA(ierr);
	} catch(char* ex) {
	  SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
	} 
	yin->valid_GPU_array = PETSC_CUDA_GPU;
      } else if (yin->valid_GPU_array == PETSC_CUDA_BOTH) {
	/* xin and yin are both valid in both places (or yin was unallocated before the earlier call to allocatecheck
	   default to copy in GPU (this is an arbitrary choice) */
	try {
	  cusp::blas::copy(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray,*((VecSeqCUDA_Container *)yin->spptr)->GPUarray);
	  ierr = WaitForGPU();CHKERRCUDA(ierr);
	} catch(char* ex) {
	  SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
	} 
	yin->valid_GPU_array = PETSC_CUDA_GPU;
      } else {
	ierr = VecCopy_Seq(xin,yin);CHKERRQ(ierr);
      }
    }
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecSwap_SeqCUDA"
PetscErrorCode VecSwap_SeqCUDA(Vec xin,Vec yin)
{
  PetscErrorCode ierr;
  PetscBLASInt   one = 1,bn = PetscBLASIntCast(xin->map->n);

  PetscFunctionBegin;
  if (xin != yin) {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
#if defined(PETSC_USE_SCALAR_SINGLE)
    hipblasSswap(bn,VecCUDACastToRawPtr(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray),one,VecCUDACastToRawPtr(*((VecSeqCUDA_Container *)yin->spptr)->GPUarray),one);
#else
    hipblasDswap(bn,VecCUDACastToRawPtr(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray),one,VecCUDACastToRawPtr(*((VecSeqCUDA_Container *)yin->spptr)->GPUarray),one);
#endif
    ierr = cublasGetError();CHKERRCUDA(ierr);
    ierr = WaitForGPU();CHKERRCUDA(ierr);
    xin->valid_GPU_array = PETSC_CUDA_GPU;
    yin->valid_GPU_array = PETSC_CUDA_GPU;
  }
  PetscFunctionReturn(0);
}

struct VecCUDAAX
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t)*thrust::get<2>(t);
  }
};
#undef __FUNCT__  
#define __FUNCT__ "VecAXPBY_SeqCUDA"
PetscErrorCode VecAXPBY_SeqCUDA(Vec yin,PetscScalar alpha,PetscScalar beta,Vec xin)
{
  PetscErrorCode    ierr;
  PetscInt          n = yin->map->n;
  PetscScalar       a = alpha,b = beta;
 
  PetscFunctionBegin;
  if (a == 0.0) {
    ierr = VecScale_SeqCUDA(yin,beta);CHKERRQ(ierr);
  } else if (b == 1.0) {
    ierr = VecAXPY_SeqCUDA(yin,alpha,xin);CHKERRQ(ierr);
  } else if (a == 1.0) {
    ierr = VecAYPX_SeqCUDA(yin,beta,xin);CHKERRQ(ierr);
  } else if (b == 0.0) {
    try {
      thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(a,0),
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->end(),  
		thrust::make_constant_iterator(a,n),
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->end())),
	VecCUDAAX());
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    ierr = PetscLogFlops(xin->map->n);CHKERRQ(ierr);
    ierr = WaitForGPU();CHKERRCUDA(ierr);
  } else {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
    try {
      cusp::blas::axpby(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray,*((VecSeqCUDA_Container *)yin->spptr)->GPUarray,*((VecSeqCUDA_Container *)yin->spptr)->GPUarray,a,b);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    yin->valid_GPU_array = PETSC_CUDA_GPU;
    ierr = WaitForGPU();CHKERRCUDA(ierr);
    ierr = PetscLogFlops(3.0*xin->map->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/* structs below are for special cases of VecAXPBYPCZ_SeqCUDA */
struct VecCUDAXPBYPCZ
{
  /* z = x + b*y + c*z */
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t)*thrust::get<0>(t)+thrust::get<2>(t)+thrust::get<4>(t)*thrust::get<3>(t);
  }
};
struct VecCUDAAXPBYPZ
{
  /* z = ax + b*y + z */
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) += thrust::get<2>(t)*thrust::get<1>(t)+thrust::get<4>(t)*thrust::get<3>(t);
  }
};

#undef __FUNCT__  
#define __FUNCT__ "VecAXPBYPCZ_SeqCUDA"
PetscErrorCode VecAXPBYPCZ_SeqCUDA(Vec zin,PetscScalar alpha,PetscScalar beta,PetscScalar gamma,Vec xin,Vec yin)
{
  PetscErrorCode     ierr;
  PetscInt           n = zin->map->n;

  PetscFunctionBegin;
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(zin);CHKERRQ(ierr);
  if (alpha == 1.0) {
    try {
      thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)zin->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(gamma,0),
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->begin(),
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(beta,0))),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)zin->spptr)->GPUarray->end(),  
		thrust::make_constant_iterator(gamma,n),
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->end(),
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->end(),
		thrust::make_constant_iterator(beta,n))),
	VecCUDAXPBYPCZ());
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr);
  } else if (gamma == 1.0) {
    try {
      thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)zin->spptr)->GPUarray->begin(),
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(alpha,0),
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->begin(),
		thrust::make_constant_iterator(beta,0))),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((VecSeqCUDA_Container*)zin->spptr)->GPUarray->end(),  
		((VecSeqCUDA_Container*)xin->spptr)->GPUarray->end(),
		thrust::make_constant_iterator(alpha,n),	
		((VecSeqCUDA_Container*)yin->spptr)->GPUarray->end(),
		thrust::make_constant_iterator(beta,n))),
	VecCUDAAXPBYPZ());
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr); 
  } else {
    try {
      cusp::blas::axpbypcz(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray,*((VecSeqCUDA_Container *)yin->spptr)->GPUarray,*((VecSeqCUDA_Container *)zin->spptr)->GPUarray,*((VecSeqCUDA_Container *)zin->spptr)->GPUarray,alpha,beta,gamma);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    zin->valid_GPU_array = PETSC_CUDA_GPU;
    ierr = PetscLogFlops(5.0*n);CHKERRQ(ierr);    
  }
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecPointwiseMult_SeqCUDA"
PetscErrorCode VecPointwiseMult_SeqCUDA(Vec win,Vec xin,Vec yin)
{
  PetscErrorCode ierr;
  PetscInt       n = win->map->n;

  PetscFunctionBegin;
  ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
  ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
  ierr = VecCUDAAllocateCheck(win);CHKERRQ(ierr);
  try {
    cusp::blas::xmy(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray,*((VecSeqCUDA_Container *)yin->spptr)->GPUarray,*((VecSeqCUDA_Container *)win->spptr)->GPUarray);
  } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
  } 
  win->valid_GPU_array = PETSC_CUDA_GPU;
  ierr = PetscLogFlops(n);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecView_SeqCUDA"
PetscErrorCode VecView_SeqCUDA(Vec xin,PetscViewer viewer)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUDACopyFromGPU(xin);CHKERRQ(ierr);
  ierr = VecView_Seq(xin,viewer);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/* should do infinity norm in cuda */

#undef __FUNCT__  
#define __FUNCT__ "VecNorm_SeqCUDA"
PetscErrorCode VecNorm_SeqCUDA(Vec xin,NormType type,PetscReal* z)
{
  PetscScalar    *xx;
  PetscErrorCode ierr;
  PetscInt       n = xin->map->n;
  PetscBLASInt   one = 1, bn = PetscBLASIntCast(n);

  PetscFunctionBegin;
  if (type == NORM_2 || type == NORM_FROBENIUS) {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    try {
      *z = cusp::blas::nrm2(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
    } 
    ierr = WaitForGPU();CHKERRCUDA(ierr);
    ierr = PetscLogFlops(PetscMax(2.0*n-1,0.0));CHKERRQ(ierr);
  } else if (type == NORM_INFINITY) {
    PetscInt     i;
    PetscReal    max = 0.0,tmp;

    ierr = VecGetArrayPrivate(xin,&xx);CHKERRQ(ierr);
    for (i=0; i<n; i++) {
      if ((tmp = PetscAbsScalar(*xx)) > max) max = tmp;
      /* check special case of tmp == NaN */
      if (tmp != tmp) {max = tmp; break;}
      xx++;
    }
    ierr = VecRestoreArrayPrivate(xin,&xx);CHKERRQ(ierr);
    *z   = max;
  } else if (type == NORM_1) {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
#if defined(PETSC_USE_SCALAR_SINGLE)
    *z = hipblasSasum(bn,VecCUDACastToRawPtr(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray),one);
#else
    *z = hipblasDasum(bn,VecCUDACastToRawPtr(*((VecSeqCUDA_Container *)xin->spptr)->GPUarray),one);
#endif
    ierr = cublasGetError();CHKERRCUDA(ierr);
    ierr = WaitForGPU();CHKERRCUDA(ierr);
    ierr = PetscLogFlops(PetscMax(n-1.0,0.0));CHKERRQ(ierr);
  } else if (type == NORM_1_AND_2) {
    ierr = VecNorm_SeqCUDA(xin,NORM_1,z);CHKERRQ(ierr);
    ierr = VecNorm_SeqCUDA(xin,NORM_2,z+1);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}


/*the following few functions should be modified to actually work with the GPU so they don't force unneccesary allocation of CPU memory */

#undef __FUNCT__  
#define __FUNCT__ "VecSetRandom_SeqCUDA"
PetscErrorCode VecSetRandom_SeqCUDA(Vec xin,PetscRandom r)
{
  PetscErrorCode ierr;
  PetscFunctionBegin;
  ierr = VecSetRandom_Seq(xin,r);CHKERRQ(ierr);
  if (xin->valid_GPU_array != PETSC_CUDA_UNALLOCATED){
    xin->valid_GPU_array = PETSC_CUDA_CPU;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecResetArray_SeqCUDA"
PetscErrorCode VecResetArray_SeqCUDA(Vec vin)
{
  PetscErrorCode ierr;
  PetscFunctionBegin;
  ierr = VecResetArray_Seq(vin);CHKERRQ(ierr);
  if (vin->valid_GPU_array != PETSC_CUDA_UNALLOCATED){
    vin->valid_GPU_array = PETSC_CUDA_CPU;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecPlaceArray_SeqCUDA"
PetscErrorCode VecPlaceArray_SeqCUDA(Vec vin,const PetscScalar *a)
{
  PetscErrorCode ierr;
  PetscFunctionBegin;
  ierr = VecPlaceArray_Seq(vin,a);CHKERRQ(ierr);
  if (vin->valid_GPU_array != PETSC_CUDA_UNALLOCATED){
    vin->valid_GPU_array = PETSC_CUDA_CPU;
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecReplaceArray_SeqCUDA"
PetscErrorCode VecReplaceArray_SeqCUDA(Vec vin,const PetscScalar *a)
{
  PetscErrorCode ierr;
  PetscFunctionBegin;
  ierr = VecReplaceArray_Seq(vin,a);CHKERRQ(ierr);
  if (vin->valid_GPU_array != PETSC_CUDA_UNALLOCATED){
    vin->valid_GPU_array = PETSC_CUDA_CPU;
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecCreateSeqCUDA"
/*@
   VecCreateSeqCUDA - Creates a standard, sequential array-style vector.

   Collective on MPI_Comm

   Input Parameter:
+  comm - the communicator, should be PETSC_COMM_SELF
-  n - the vector length 

   Output Parameter:
.  V - the vector

   Notes:
   Use VecDuplicate() or VecDuplicateVecs() to form additional vectors of the
   same type as an existing vector.

   Level: intermediate

   Concepts: vectors^creating sequential

.seealso: VecCreateMPI(), VecCreate(), VecDuplicate(), VecDuplicateVecs(), VecCreateGhost()
@*/
PetscErrorCode PETSCVEC_DLLEXPORT VecCreateSeqCUDA(MPI_Comm comm,PetscInt n,Vec *v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCreate(comm,v);CHKERRQ(ierr);
  ierr = VecSetSizes(*v,n,n);CHKERRQ(ierr);
  ierr = VecSetType(*v,VECSEQCUDA);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*The following template functions are for VecDotNorm2_SeqCUDA.  Note that there is no complex support as currently written*/
template <typename T>
struct cudadotnormcalculate : thrust::unary_function<T,T>
{
	__host__ __device__
	T operator()(T x)
	{
		return thrust::make_tuple(thrust::get<0>(x)*thrust::get<1>(x),thrust::get<1>(x)*thrust::get<1>(x));
	}
};

template <typename T>
struct cudadotnormreduce : thrust::binary_function<T,T,T>
{
	__host__ __device__
	T operator()(T x,T y)
	{
		return thrust::make_tuple(thrust::get<0>(x)+thrust::get<0>(y),thrust::get<1>(x)+thrust::get<1>(y));
	}
};
	
#undef __FUNCT__
#define __FUNCT__ "VecDotNorm2_SeqCUDA"
PetscErrorCode VecDotNorm2_SeqCUDA(Vec s, Vec t, PetscScalar *dp, PetscScalar *nm)
{
  PetscErrorCode                         ierr;
  PetscScalar                            zero = 0.0,n=s->map->n;
  thrust::tuple<PetscScalar,PetscScalar> result;

  PetscFunctionBegin;
  ierr = VecCUDACopyToGPU(s);CHKERRQ(ierr);
  ierr = VecCUDACopyToGPU(t);CHKERRQ(ierr);
  try {
    result = thrust::transform_reduce(
		 thrust::make_zip_iterator(
		     thrust::make_tuple(
			 ((VecSeqCUDA_Container *)s->spptr)->GPUarray->begin(),
			 ((VecSeqCUDA_Container *)t->spptr)->GPUarray->begin())),
		 thrust::make_zip_iterator(
                     thrust::make_tuple(
			 ((VecSeqCUDA_Container *)s->spptr)->GPUarray->end(),
			 ((VecSeqCUDA_Container *)t->spptr)->GPUarray->end())),
		  cudadotnormcalculate<thrust::tuple<PetscScalar,PetscScalar> >(),
		  thrust::make_tuple(zero,zero), /*init */
		  cudadotnormreduce<thrust::tuple<PetscScalar, PetscScalar> >()); /* binary function */
    *dp = thrust::get<0>(result);
    *nm = thrust::get<1>(result);
  } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
  } 
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecDuplicate_SeqCUDA"
PetscErrorCode VecDuplicate_SeqCUDA(Vec win,Vec *V)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCreateSeqCUDA(((PetscObject)win)->comm,win->map->n,V);CHKERRQ(ierr);
  if (win->mapping) {
    ierr = PetscObjectReference((PetscObject)win->mapping);CHKERRQ(ierr);
    (*V)->mapping = win->mapping;
  }
  if (win->bmapping) {
    ierr = PetscObjectReference((PetscObject)win->bmapping);CHKERRQ(ierr);
    (*V)->bmapping = win->bmapping;
  }
  (*V)->map->bs = win->map->bs;
  ierr = PetscOListDuplicate(((PetscObject)win)->olist,&((PetscObject)(*V))->olist);CHKERRQ(ierr);
  ierr = PetscFListDuplicate(((PetscObject)win)->qlist,&((PetscObject)(*V))->qlist);CHKERRQ(ierr);

  (*V)->stash.ignorenegidx = win->stash.ignorenegidx;
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecDestroy_SeqCUDA"
PetscErrorCode VecDestroy_SeqCUDA(Vec v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  try {
    if (v->spptr) {
      if (((VecSeqCUDA_Container *)v->spptr)->GPUarray) {
	delete ((VecSeqCUDA_Container *)v->spptr)->GPUarray;
      }
      delete (VecSeqCUDA_Container *)v->spptr;
    }
  } catch(char* ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUDA error: %s", ex);
  } 
  ierr = VecDestroy_Seq(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

EXTERN_C_BEGIN
#undef __FUNCT__  
#define __FUNCT__ "VecCreate_SeqCUDA"
PetscErrorCode PETSCVEC_DLLEXPORT VecCreate_SeqCUDA(Vec V)
{
  PetscErrorCode ierr;
  PetscMPIInt    size;
 
  PetscFunctionBegin;
  ierr = MPI_Comm_size(((PetscObject)V)->comm,&size);CHKERRQ(ierr);
  if  (size > 1) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Cannot create VECSEQCUDA on more than one process");
  ierr = VecCreate_Seq_Private(V,0);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)V,VECSEQCUDA);CHKERRQ(ierr);
  V->ops->dot             = VecDot_SeqCUDA;
  V->ops->norm            = VecNorm_SeqCUDA;
  V->ops->tdot            = VecTDot_SeqCUDA;
  V->ops->scale           = VecScale_SeqCUDA;
  V->ops->copy            = VecCopy_SeqCUDA;
  V->ops->set             = VecSet_SeqCUDA;
  V->ops->swap            = VecSwap_SeqCUDA;
  V->ops->axpy            = VecAXPY_SeqCUDA;
  V->ops->axpby           = VecAXPBY_SeqCUDA;
  V->ops->axpbypcz        = VecAXPBYPCZ_SeqCUDA;
  V->ops->pointwisemult   = VecPointwiseMult_SeqCUDA;
  V->ops->pointwisedivide = VecPointwiseDivide_SeqCUDA;
  V->ops->setrandom       = VecSetRandom_SeqCUDA;
  V->ops->view            = VecView_SeqCUDA;
  V->ops->dot_local       = VecDot_SeqCUDA;
  V->ops->tdot_local      = VecTDot_SeqCUDA;
  V->ops->norm_local      = VecNorm_SeqCUDA;
  V->ops->maxpy           = VecMAXPY_SeqCUDA;
  V->ops->mdot            = VecMDot_SeqCUDA;
  V->ops->aypx            = VecAYPX_SeqCUDA;
  V->ops->waxpy           = VecWAXPY_SeqCUDA;
  V->ops->dotnorm2        = VecDotNorm2_SeqCUDA;
  V->ops->placearray      = VecPlaceArray_SeqCUDA;
  V->ops->replacearray    = VecReplaceArray_SeqCUDA;
  V->ops->resetarray      = VecResetArray_SeqCUDA;
  V->ops->destroy         = VecDestroy_SeqCUDA;
  V->ops->duplicate       = VecDuplicate_SeqCUDA;
  V->valid_GPU_array      = PETSC_CUDA_UNALLOCATED;
  PetscFunctionReturn(0);
}
EXTERN_C_END
