#include "hip/hip_runtime.h"
#define PETSCVEC_DLL
/*
   Implements the sequential vectors.
*/

#include "petscconf.h"
PETSC_CUDA_EXTERN_C_BEGIN
#include "private/vecimpl.h"          /*I "petscvec.h" I*/
#include "../src/vec/vec/impls/dvecimpl.h"
PETSC_CUDA_EXTERN_C_END
#include "../src/vec/vec/impls/seq/seqcuda/cudavecimpl.h"
/* these following 2 public versions are necessary because we use CUSP in the regular version and these need to be called from plain C code. */
#undef __FUNCT__
#define __FUNCT__ "VecCUDAAllocateCheck_Public"
PetscErrorCode VecCUDAAllocateCheck_Public(Vec v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUDAAllocateCheck(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUDACopyToGPU_Public"
PetscErrorCode VecCUDACopyToGPU_Public(Vec v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUDACopyToGPU(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCUDACopyFromGPU"
/* Copies a vector from the GPU to the CPU unless we already have an up-to-date copy on the CPU */
PetscErrorCode VecCUDACopyFromGPU(Vec v)
{
  PetscErrorCode ierr;
  CUSPARRAY      *GPUvector = (CUSPARRAY *)(v->spptr);
  PetscScalar    *array;
  Vec_Seq        *s;
  PetscInt       n = v->map->n;

  PetscFunctionBegin;
  s = (Vec_Seq*)v->data;
  if (s->array == 0){
    ierr               = PetscMalloc(n*sizeof(PetscScalar),&array);CHKERRQ(ierr);
    ierr               = PetscLogObjectMemory(v,n*sizeof(PetscScalar));CHKERRQ(ierr);
    s->array           = array;
    s->array_allocated = array;
  }
  if (v->valid_GPU_array == PETSC_CUDA_GPU){
    ierr = PetscLogEventBegin(VEC_CUDACopyFromGPU,v,0,0,0);CHKERRQ(ierr);
    thrust::copy(GPUvector->begin(),GPUvector->end(),*(PetscScalar**)v->data);
    ierr = PetscLogEventEnd(VEC_CUDACopyFromGPU,v,0,0,0);CHKERRQ(ierr);
    v->valid_GPU_array = PETSC_CUDA_BOTH;
  }
  PetscFunctionReturn(0);
}





/*MC
   VECSEQCUDA - VECSEQCUDA = "seqcuda" - The basic sequential vector, modified to use CUDA

   Options Database Keys:
. -vec_type seqcuda - sets the vector type to VECSEQCUDA during a call to VecSetFromOptions()

  Level: beginner

.seealso: VecCreate(), VecSetType(), VecSetFromOptions(), VecCreateSeqWithArray(), VECMPI, VecType, VecCreateMPI(), VecCreateSeq()
M*/

/* for VecAYPX_SeqCUDA*/
namespace cusp
{
namespace blas
{
namespace detail
{
  template <typename T>
    struct AYPX : public thrust::binary_function<T,T,T>
    {
      T alpha;
      
      AYPX(T _alpha) : alpha(_alpha) {}

      __host__ __device__
	T operator()(T x, T y)
      {
	return alpha * y + x;
      }
    };
}

 template <typename ForwardIterator1,
           typename ForwardIterator2,
           typename ScalarType>
void aypx(ForwardIterator1 first1,ForwardIterator1 last1,ForwardIterator2 first2,ScalarType alpha)
	   {
	     thrust::transform(first1,last1,first2,first2,detail::AYPX<ScalarType>(alpha));
	   }
 template <typename Array1, typename Array2, typename ScalarType>
   void aypx(const Array1& x, Array2& y, ScalarType alpha)
 {
   detail::assert_same_dimensions(x,y);
   aypx(x.begin(),x.end(),y.begin(),alpha);
 }
}
}

#undef __FUNCT__
#define __FUNCT__ "VecAYPX_SeqCUDA"
PetscErrorCode VecAYPX_SeqCUDA(Vec yin, PetscScalar alpha, Vec xin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (alpha != 0.0) {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
    cusp::blas::aypx(*(CUSPARRAY *)(xin->spptr),*(CUSPARRAY *)(yin->spptr),alpha);
    yin->valid_GPU_array = PETSC_CUDA_GPU;
    ierr = PetscLogFlops(2.0*yin->map->n);CHKERRQ(ierr);
   }
  PetscFunctionReturn(0);
}

     

#undef __FUNCT__  
#define __FUNCT__ "VecAXPY_SeqCUDA"
PetscErrorCode VecAXPY_SeqCUDA(Vec yin,PetscScalar alpha,Vec xin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  /* assume that the BLAS handles alpha == 1.0 efficiently since we have no fast code for it */
  if (alpha != 0.0) {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
    cusp::blas::axpy(*(CUSPARRAY *)(xin->spptr),*(CUSPARRAY *)(yin->spptr),alpha);
    yin->valid_GPU_array = PETSC_CUDA_GPU;
    ierr = PetscLogFlops(2.0*yin->map->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

struct VecCUDAPointwiseDivide
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t) / thrust::get<2>(t);
  }
};

#undef __FUNCT__
#define __FUNCT__ "VecPointwiseDivide_SeqCUDA"
PetscErrorCode VecPointwiseDivide_SeqCUDA(Vec win, Vec xin, Vec yin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUDAAllocateCheck(win);CHKERRQ(ierr);
  ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
  ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
  thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)win->spptr)->begin(),
		((CUSPARRAY*)xin->spptr)->begin(),
		((CUSPARRAY*)yin->spptr)->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)win->spptr)->end(),  
		((CUSPARRAY*)xin->spptr)->end(),
		((CUSPARRAY*)yin->spptr)->end())),
	VecCUDAPointwiseDivide());
  ierr = PetscLogFlops(win->map->n);CHKERRQ(ierr);
  win->valid_GPU_array = PETSC_CUDA_GPU;
  PetscFunctionReturn(0);
}


struct VecCUDAWAXPY
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t) + thrust::get<2>(t)*thrust::get<3>(t);
  }
};

struct VecCUDASum
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t) + thrust::get<2>(t);
  }
};

struct VecCUDADiff
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t) - thrust::get<2>(t);
  }
};

#undef __FUNCT__
#define __FUNCT__ "VecWAXPY_SeqCUDA"
PetscErrorCode VecWAXPY_SeqCUDA(Vec win,PetscScalar alpha,Vec xin, Vec yin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
  ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
  ierr = VecCUDAAllocateCheck(win);CHKERRQ(ierr);
  if (alpha == 1.0) {
    thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)win->spptr)->begin(),
		((CUSPARRAY*)yin->spptr)->begin(),
		((CUSPARRAY*)xin->spptr)->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)win->spptr)->end(),  
		((CUSPARRAY*)yin->spptr)->end(),
		((CUSPARRAY*)xin->spptr)->end())),
	VecCUDASum());
    ierr = PetscLogFlops(win->map->n);CHKERRQ(ierr);
  } else if (alpha == -1.0) {
     thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)win->spptr)->begin(),
		((CUSPARRAY*)yin->spptr)->begin(),
		((CUSPARRAY*)xin->spptr)->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)win->spptr)->end(),  
		((CUSPARRAY*)yin->spptr)->end(),
		((CUSPARRAY*)xin->spptr)->end())),
	VecCUDADiff());
     ierr = PetscLogFlops(win->map->n);CHKERRQ(ierr);
  } else if (alpha == 0.0) {
    ierr = VecCopy_SeqCUDA(yin,win);CHKERRQ(ierr);
  } else {
     thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)win->spptr)->begin(),
		((CUSPARRAY*)yin->spptr)->begin(),
		thrust::make_constant_iterator(alpha,0),
		((CUSPARRAY*)xin->spptr)->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)win->spptr)->end(),  
		((CUSPARRAY*)yin->spptr)->end(),
		thrust::make_constant_iterator(alpha,win->map->n),
		((CUSPARRAY*)xin->spptr)->end())),
	VecCUDAWAXPY());
     ierr = PetscLogFlops(2*win->map->n);CHKERRQ(ierr);
  }
  win->valid_GPU_array = PETSC_CUDA_GPU;
  PetscFunctionReturn(0);
}

/* These functions are for the CUDA implementation of MAXPY with the loop unrolled on the CPU */
struct VecCUDAMAXPY4
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    /*y += a1*x1 +a2*x2 + 13*x3 +a4*x4 */
    thrust::get<0>(t) += thrust::get<1>(t)*thrust::get<2>(t)+thrust::get<3>(t)*thrust::get<4>(t)+thrust::get<5>(t)*thrust::get<6>(t)+thrust::get<7>(t)*thrust::get<8>(t);
  }
};


struct VecCUDAMAXPY3
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    /*y += a1*x1 +a2*x2 + 13*x3 */
    thrust::get<0>(t) += thrust::get<1>(t)*thrust::get<2>(t)+thrust::get<3>(t)*thrust::get<4>(t)+thrust::get<5>(t)*thrust::get<6>(t);
  }
};

struct VecCUDAMAXPY2
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    /*y += a1*x1 +a2*x2*/
    thrust::get<0>(t) += thrust::get<1>(t)*thrust::get<2>(t)+thrust::get<3>(t)*thrust::get<4>(t);
  }
};
#undef __FUNCT__  
#define __FUNCT__ "VecMAXPY_SeqCUDA"
PetscErrorCode VecMAXPY_SeqCUDA(Vec xin, PetscInt nv,const PetscScalar *alpha,Vec *y)
{
  PetscErrorCode    ierr;
  PetscInt          n = xin->map->n,j,j_rem;
  Vec               yy0,yy1,yy2,yy3;
  PetscScalar       alpha0,alpha1,alpha2,alpha3;

  PetscFunctionBegin;
  ierr = PetscLogFlops(nv*2.0*n);CHKERRQ(ierr);
  ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
  switch (j_rem=nv&0x3) {
  case 3: 
    alpha0 = alpha[0]; 
    alpha1 = alpha[1]; 
    alpha2 = alpha[2]; 
    alpha += 3;
    yy0    = y[0];
    yy1    = y[1];
    yy2    = y[2];
    ierr   = VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr   = VecCUDACopyToGPU(yy1);CHKERRQ(ierr);
    ierr   = VecCUDACopyToGPU(yy2);CHKERRQ(ierr);
    thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)xin->spptr)->begin(),
		thrust::make_constant_iterator(alpha0,0),
		((CUSPARRAY*)yy0->spptr)->begin(),
		thrust::make_constant_iterator(alpha1,0),
		((CUSPARRAY*)yy1->spptr)->begin(),
		thrust::make_constant_iterator(alpha2,0),
		((CUSPARRAY*)yy2->spptr)->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)xin->spptr)->end(),  
		thrust::make_constant_iterator(alpha0,n),
		((CUSPARRAY*)yy0->spptr)->end(),
		thrust::make_constant_iterator(alpha1,n),
		((CUSPARRAY*)yy1->spptr)->end(),
		thrust::make_constant_iterator(alpha2,n),
		((CUSPARRAY*)yy2->spptr)->end())),
	VecCUDAMAXPY3());
    y     += 3;
    break;
  case 2: 
    alpha0 = alpha[0]; 
    alpha1 = alpha[1]; 
    alpha +=2;
    yy0    = y[0];
    yy1    = y[1];
    ierr   = VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr   = VecCUDACopyToGPU(yy1);CHKERRQ(ierr);
    thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)xin->spptr)->begin(),
		thrust::make_constant_iterator(alpha0,0),
		((CUSPARRAY*)yy0->spptr)->begin(),
		thrust::make_constant_iterator(alpha1,0),
		((CUSPARRAY*)yy1->spptr)->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)xin->spptr)->end(),  
		thrust::make_constant_iterator(alpha0,n),
		((CUSPARRAY*)yy0->spptr)->end(),
		thrust::make_constant_iterator(alpha1,n),
		((CUSPARRAY*)yy1->spptr)->end())),
	VecCUDAMAXPY2());
    y     +=2;
    break;
  case 1: 
    alpha0 = *alpha++; 
    yy0 = y[0];
    ierr =  VecAXPY_SeqCUDA(xin,alpha0,yy0);
    y     +=1;
    break;
  }
  for (j=j_rem; j<nv; j+=4) {
    alpha0 = alpha[0];
    alpha1 = alpha[1];
    alpha2 = alpha[2];
    alpha3 = alpha[3];
    alpha  += 4;
    yy0    = y[0];
    yy1    = y[1];
    yy2    = y[2];
    yy3    = y[3];
    ierr   = VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr   = VecCUDACopyToGPU(yy1);CHKERRQ(ierr);
    ierr   = VecCUDACopyToGPU(yy2);CHKERRQ(ierr);
    ierr   = VecCUDACopyToGPU(yy3);CHKERRQ(ierr);
    thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)xin->spptr)->begin(),
		thrust::make_constant_iterator(alpha0,0),
		((CUSPARRAY*)yy0->spptr)->begin(),
		thrust::make_constant_iterator(alpha1,0),
		((CUSPARRAY*)yy1->spptr)->begin(),
		thrust::make_constant_iterator(alpha2,0),
		((CUSPARRAY*)yy2->spptr)->begin(),
		thrust::make_constant_iterator(alpha3,0),
		((CUSPARRAY*)yy3->spptr)->begin())),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)xin->spptr)->end(),  
		thrust::make_constant_iterator(alpha0,n),
		((CUSPARRAY*)yy0->spptr)->end(),
		thrust::make_constant_iterator(alpha1,n),
		((CUSPARRAY*)yy1->spptr)->end(),
		thrust::make_constant_iterator(alpha2,n),
		((CUSPARRAY*)yy2->spptr)->end(),
		thrust::make_constant_iterator(alpha3,n),
		((CUSPARRAY*)yy3->spptr)->end())),
	VecCUDAMAXPY4());
    y      += 4;
  }
  xin->valid_GPU_array = PETSC_CUDA_GPU;
  PetscFunctionReturn(0);
} 


#undef __FUNCT__
#define __FUNCT__ "VecDot_SeqCUDA"
PetscErrorCode VecDot_SeqCUDA(Vec xin,Vec yin,PetscScalar *z)
{
#if defined(PETSC_USE_COMPLEX)
  PetscScalar    *ya,*xa;
#endif
  PetscErrorCode ierr;
  PetscFunctionBegin;
#if defined(PETSC_USE_COMPLEX)
  /* cannot use BLAS dot for complex because compiler/linker is 
     not happy about returning a double complex */
  {
    ierr = VecGetArrayPrivate2(xin,&xa,yin,&ya);CHKERRQ(ierr);
    PetscInt    i;
    PetscScalar sum = 0.0;
    for (i=0; i<xin->map->n; i++) {
      sum += xa[i]*PetscConj(ya[i]);
    }
    *z = sum;
    ierr = VecRestoreArrayPrivate2(xin,&xa,yin,&ya);CHKERRQ(ierr);
  }
#else
  {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
    *z = cusp::blas::dot(*(CUSPARRAY *)(xin->spptr),*(CUSPARRAY *)(yin->spptr));
  }
#endif
  if (xin->map->n >0) {
    ierr = PetscLogFlops(2.0*xin->map->n-1);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*The following few template functions are for VecMDot_SeqCUDA*/

template <typename T1,typename T2>
struct cudamult2 : thrust::unary_function<T1,T2>
{
	__host__ __device__
	T2 operator()(T1 x)
	{
		return thrust::make_tuple(thrust::get<0>(x)*thrust::get<1>(x),thrust::get<0>(x)*thrust::get<2>(x));
	}
};

template <typename T>
struct cudaadd2 : thrust::binary_function<T,T,T>
{
	__host__ __device__
	T operator()(T x,T y)
	{
		return thrust::make_tuple(thrust::get<0>(x)+thrust::get<0>(y),thrust::get<1>(x)+thrust::get<1>(y));
	}
};
	
template <typename T1,typename T2>
struct cudamult3 : thrust::unary_function<T1,T2>
{
	__host__ __device__
	T2 operator()(T1 x)
	{
	  return thrust::make_tuple(thrust::get<0>(x)*thrust::get<1>(x),thrust::get<0>(x)*thrust::get<2>(x),thrust::get<0>(x)*thrust::get<3>(x));
	}
};

template <typename T>
struct cudaadd3 : thrust::binary_function<T,T,T>
{
	__host__ __device__
	T operator()(T x,T y)
	{
	  return thrust::make_tuple(thrust::get<0>(x)+thrust::get<0>(y),thrust::get<1>(x)+thrust::get<1>(y),thrust::get<2>(x)+thrust::get<2>(y));
	}
};
	template <typename T1,typename T2>
struct cudamult4 : thrust::unary_function<T1,T2>
{
	__host__ __device__
	T2 operator()(T1 x)
	{
	  return thrust::make_tuple(thrust::get<0>(x)*thrust::get<1>(x),thrust::get<0>(x)*thrust::get<2>(x),thrust::get<0>(x)*thrust::get<3>(x),thrust::get<0>(x)*thrust::get<4>(x));
	}
};

template <typename T>
struct cudaadd4 : thrust::binary_function<T,T,T>
{
	__host__ __device__
	T operator()(T x,T y)
	{
	  return thrust::make_tuple(thrust::get<0>(x)+thrust::get<0>(y),thrust::get<1>(x)+thrust::get<1>(y),thrust::get<2>(x)+thrust::get<2>(y),thrust::get<3>(x)+thrust::get<3>(y));
	}
};


#undef __FUNCT__  
#define __FUNCT__ "VecMDot_SeqCUDA"
PetscErrorCode VecMDot_SeqCUDA(Vec xin,PetscInt nv,const Vec yin[],PetscScalar *z)
{
  PetscErrorCode    ierr;
  PetscInt          n = xin->map->n,j,j_rem;
  Vec               yy0,yy1,yy2,yy3;
  PetscScalar       zero=0.0;
  thrust::tuple<PetscScalar,PetscScalar> result2;
  thrust::tuple<PetscScalar,PetscScalar,PetscScalar> result3;
  thrust::tuple<PetscScalar,PetscScalar,PetscScalar,PetscScalar>result4;

  PetscFunctionBegin;
  ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
  switch(j_rem=nv&0x3) {
  case 3: 
    yy0  =  yin[0];
    yy1  =  yin[1];
    yy2  =  yin[2];
    ierr =  VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr =  VecCUDACopyToGPU(yy1);CHKERRQ(ierr);
    ierr =  VecCUDACopyToGPU(yy2);CHKERRQ(ierr);
    result3 = thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(((CUSPARRAY *)xin->spptr)->begin(),((CUSPARRAY *)yy0->spptr)->begin(),((CUSPARRAY *)yy1->spptr)->begin(), ((CUSPARRAY *)yy2->spptr)->begin())),
				       thrust::make_zip_iterator(thrust::make_tuple(((CUSPARRAY *)xin->spptr)->end(),((CUSPARRAY *)yy0->spptr)->end(),((CUSPARRAY *)yy1->spptr)->end(),((CUSPARRAY *)yy2->spptr)->end())),
				       cudamult3<thrust::tuple<PetscScalar,PetscScalar,PetscScalar,PetscScalar>, thrust::tuple<PetscScalar,PetscScalar,PetscScalar> >(),
				       thrust::make_tuple(zero,zero,zero), /*init */
				       cudaadd3<thrust::tuple<PetscScalar,PetscScalar,PetscScalar> >()); /* binary function */
    z[0] = thrust::get<0>(result3);
    z[1] = thrust::get<1>(result3);
    z[2] = thrust::get<2>(result3);
    
    z    += 3;
    yin  += 3;
    break;
  case 2:
    yy0  =  yin[0];
    yy1  =  yin[1];
    ierr =  VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr =  VecCUDACopyToGPU(yy1);CHKERRQ(ierr);
    result2 = thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(((CUSPARRAY *)xin->spptr)->begin(),((CUSPARRAY *)yy0->spptr)->begin(),((CUSPARRAY *)yy1->spptr)->begin())),
				    thrust::make_zip_iterator(thrust::make_tuple(((CUSPARRAY *)xin->spptr)->end(),((CUSPARRAY *)yy0->spptr)->end(),((CUSPARRAY *)yy1->spptr)->end())),
				    cudamult2<thrust::tuple<PetscScalar,PetscScalar,PetscScalar>, thrust::tuple<PetscScalar,PetscScalar> >(),
				    thrust::make_tuple(zero,zero), /*init */
				    cudaadd2<thrust::tuple<PetscScalar, PetscScalar> >()); /* binary function */
    z[0] = thrust::get<0>(result2);
    z[1] = thrust::get<1>(result2);
    

    z    += 2;
    yin  += 2;
    break;
  case 1: 
    yy0  =  yin[0];
    ierr =  VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr =  VecDot_SeqCUDA(xin,yy0,&z[0]);CHKERRQ(ierr);
    z    += 1;
    yin  += 1;
    break;
  }
  for (j=j_rem; j<nv; j+=4) {
    yy0  =  yin[0];
    yy1  =  yin[1];
    yy2  =  yin[2];
    yy3  =  yin[3];
    ierr =  VecCUDACopyToGPU(yy0);CHKERRQ(ierr);
    ierr =  VecCUDACopyToGPU(yy1);CHKERRQ(ierr);
    ierr =  VecCUDACopyToGPU(yy2);CHKERRQ(ierr);
    ierr =  VecCUDACopyToGPU(yy3);CHKERRQ(ierr);
    result4 = thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(((CUSPARRAY *)xin->spptr)->begin(),((CUSPARRAY *)yy0->spptr)->begin(),((CUSPARRAY *)yy1->spptr)->begin(), ((CUSPARRAY *)yy2->spptr)->begin(),((CUSPARRAY *)yy3->spptr)->begin())),
				       thrust::make_zip_iterator(thrust::make_tuple(((CUSPARRAY *)xin->spptr)->end(),((CUSPARRAY *)yy0->spptr)->end(),((CUSPARRAY *)yy1->spptr)->end(),((CUSPARRAY *)yy2->spptr)->end(),((CUSPARRAY *)yy3->spptr)->end())),
				       cudamult4<thrust::tuple<PetscScalar,PetscScalar,PetscScalar,PetscScalar,PetscScalar>, thrust::tuple<PetscScalar,PetscScalar,PetscScalar,PetscScalar> >(),
				       thrust::make_tuple(zero,zero,zero,zero), /*init */
				       cudaadd4<thrust::tuple<PetscScalar,PetscScalar,PetscScalar,PetscScalar> >()); /* binary function */
    z[0] = thrust::get<0>(result4);
    z[1] = thrust::get<1>(result4);
    z[2] = thrust::get<2>(result4);
    z[3] = thrust::get<3>(result4);

    z    += 4;
    yin  += 4;
  }  
  ierr = PetscLogFlops(PetscMax(nv*(2.0*n-1),0.0));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecSet_SeqCUDA"
PetscErrorCode VecSet_SeqCUDA(Vec xin,PetscScalar alpha)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  /* if there's a faster way to do the case alpha=0.0 on the GPU we should do that*/
  ierr = VecCUDAAllocateCheck(xin);CHKERRQ(ierr);
  cusp::blas::fill(*(CUSPARRAY *)(xin->spptr),alpha);
  xin->valid_GPU_array = PETSC_CUDA_GPU;
  PetscFunctionReturn(0);
}

#undef __FUNCT__ 
#define __FUNCT__ "VecScale_SeqCUDA"
PetscErrorCode VecScale_SeqCUDA(Vec xin, PetscScalar alpha)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (alpha == 0.0) {
    ierr = VecSet_SeqCUDA(xin,alpha);CHKERRQ(ierr);
  } else if (alpha != 1.0) {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    cusp::blas::scal(*(CUSPARRAY *)(xin->spptr),alpha);
    xin->valid_GPU_array = PETSC_CUDA_GPU;
  }
  ierr = PetscLogFlops(xin->map->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecTDot_SeqCUDA"
PetscErrorCode VecTDot_SeqCUDA(Vec xin,Vec yin,PetscScalar *z)
{
#if defined(PETSC_USE_COMPLEX)
  PetscScalar    *ya,*xa;
#endif
  PetscErrorCode ierr;

  PetscFunctionBegin;
#if defined(PETSC_USE_COMPLEX)
  /* cannot use BLAS dot for complex because compiler/linker is 
     not happy about returning a double complex */
 ierr = VecGetArrayPrivate2(xin,&xa,yin,&ya);CHKERRQ(ierr);
 {
   PetscInt    i;
   PetscScalar sum = 0.0;
   for (i=0; i<xin->map->n; i++) {
     sum += xa[i]*ya[i];
   }
   *z = sum;
   ierr = VecRestoreArrayPrivate2(xin,&xa,yin,&ya);CHKERRQ(ierr);
 }
#else
 ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
 ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
 *z = cusp::blas::dot(*(CUSPARRAY *)(xin->spptr),*(CUSPARRAY *)(yin->spptr));
#endif
  if (xin->map->n > 0) {
    ierr = PetscLogFlops(2.0*xin->map->n-1);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}
#undef __FUNCT__  
#define __FUNCT__ "VecCopy_SeqCUDA"
PetscErrorCode VecCopy_SeqCUDA(Vec xin,Vec yin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (xin != yin) {
    if (xin->valid_GPU_array == PETSC_CUDA_GPU) {
      /* copy in GPU */
       ierr = VecCUDAAllocateCheck(yin);CHKERRQ(ierr);
      cusp::blas::copy(*(CUSPARRAY *)(xin->spptr),*(CUSPARRAY *)(yin->spptr));
      yin->valid_GPU_array = PETSC_CUDA_GPU;

    } else if (xin->valid_GPU_array == PETSC_CUDA_CPU || xin->valid_GPU_array == PETSC_CUDA_UNALLOCATED) {
      /* copy in CPU if we are on the CPU*/
      ierr = VecCopy_Seq(xin,yin);CHKERRQ(ierr);
    
    } else if (xin->valid_GPU_array == PETSC_CUDA_BOTH) {
      /* if xin is valid in both places, see where yin is and copy there (because it's probably where we'll want to next use it) */
      if (yin->valid_GPU_array == PETSC_CUDA_CPU) {
	/* copy in CPU */
	ierr = VecCopy_Seq(xin,yin);CHKERRQ(ierr);

      } else if (yin->valid_GPU_array == PETSC_CUDA_GPU) {
	/* copy in GPU */
	ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
	cusp::blas::copy(*(CUSPARRAY *)(xin->spptr),*(CUSPARRAY *)(yin->spptr));
	yin->valid_GPU_array = PETSC_CUDA_GPU;
      } else if (yin->valid_GPU_array == PETSC_CUDA_BOTH) {
	/* xin and yin are both valid in both places (or yin was unallocated before the earlier call to allocatecheck
	   default to copy in GPU (this is an arbitrary choice) */
	cusp::blas::copy(*(CUSPARRAY *)(xin->spptr),*(CUSPARRAY *)(yin->spptr));
	yin->valid_GPU_array = PETSC_CUDA_GPU;
      } else {
	ierr = VecCopy_Seq(xin,yin);CHKERRQ(ierr);
      }
    }
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecSwap_SeqCUDA"
PetscErrorCode VecSwap_SeqCUDA(Vec xin,Vec yin)
{
  PetscErrorCode ierr;
  PetscBLASInt   one = 1,bn = PetscBLASIntCast(xin->map->n);

  PetscFunctionBegin;
  if (xin != yin) {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
#if defined(PETSC_USE_SCALAR_SINGLE)
    hipblasSswap(bn,VecCUDACastToRawPtr(*(CUSPARRAY *)(xin->spptr)),one,VecCUDACastToRawPtr(*(CUSPARRAY *)(yin->spptr)),one);
#else
    hipblasDswap(bn,VecCUDACastToRawPtr(*(CUSPARRAY *)(xin->spptr)),one,VecCUDACastToRawPtr(*(CUSPARRAY *)(yin->spptr)),one);
#endif
    ierr = cublasGetError();CHKERRCUDA(ierr);
    xin->valid_GPU_array = PETSC_CUDA_GPU;
    yin->valid_GPU_array = PETSC_CUDA_GPU;
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecAXPBY_SeqCUDA"
PetscErrorCode VecAXPBY_SeqCUDA(Vec yin,PetscScalar alpha,PetscScalar beta,Vec xin)
{
  PetscErrorCode    ierr;
  PetscInt          n = yin->map->n,i;
  const PetscScalar *xx;
  PetscScalar       *yy,a = alpha,b = beta;
 
  PetscFunctionBegin;
  if (a == 0.0) {
    ierr = VecScale_SeqCUDA(yin,beta);CHKERRQ(ierr);
  } else if (b == 1.0) {
    ierr = VecAXPY_SeqCUDA(yin,alpha,xin);CHKERRQ(ierr);
  } else if (a == 1.0) {
    ierr = VecAYPX_SeqCUDA(yin,beta,xin);CHKERRQ(ierr);
  } else if (b == 0.0) {
    ierr = VecGetArrayPrivate2(xin,(PetscScalar**)&xx,yin,&yy);CHKERRQ(ierr);
    for (i=0; i<n; i++) {
      yy[i] = a*xx[i];
    }
    ierr = VecRestoreArrayPrivate2(xin,(PetscScalar**)&xx,yin,&yy);CHKERRQ(ierr);
    ierr = PetscLogFlops(xin->map->n);CHKERRQ(ierr);
  } else {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
    cusp::blas::axpby(*(CUSPARRAY *)(xin->spptr),*(CUSPARRAY *)(yin->spptr),*(CUSPARRAY *)(yin->spptr),a,b);
    yin->valid_GPU_array = PETSC_CUDA_GPU;
    ierr = PetscLogFlops(3.0*xin->map->n);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/* structs below are for special cases of VecAXPBYPCZ_SeqCUDA */
struct VecCUDAXPBYPCZ
{
  /* z = x + b*y + c*z */
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t)*thrust::get<0>(t)+thrust::get<2>(t)+thrust::get<4>(t)*thrust::get<3>(t);
  }
};
struct VecCUDAAXPBYPZ
{
  /* z = ax + b*y + z */
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) += thrust::get<2>(t)*thrust::get<1>(t)+thrust::get<4>(t)*thrust::get<3>(t);
  }
};

#undef __FUNCT__  
#define __FUNCT__ "VecAXPBYPCZ_SeqCUDA"
PetscErrorCode VecAXPBYPCZ_SeqCUDA(Vec zin,PetscScalar alpha,PetscScalar beta,PetscScalar gamma,Vec xin,Vec yin)
{
  PetscErrorCode     ierr;
  PetscInt           n = zin->map->n;

  PetscFunctionBegin;
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
    ierr = VecCUDACopyToGPU(zin);CHKERRQ(ierr);
    /*
  if (alpha == 1.0) {
    thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)zin->spptr)->begin(),
		thrust::make_constant_iterator(gamma,0),
		((CUSPARRAY*)xin->spptr)->begin(),
		((CUSPARRAY*)yin->spptr)->begin(),
		thrust::make_constant_iterator(beta,0))),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)zin->spptr)->end(),  
		thrust::make_constant_iterator(gamma,n),
		((CUSPARRAY*)xin->spptr)->end(),
		((CUSPARRAY*)yin->spptr)->end(),
		thrust::make_constant_iterator(beta,n))),
	VecCUDAXPBYPCZ());
    ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr);
  } else if (gamma == 1.0) {
    thrust::for_each(
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)zin->spptr)->begin(),
		((CUSPARRAY*)xin->spptr)->begin(),
		thrust::make_constant_iterator(alpha,0),
		((CUSPARRAY*)yin->spptr)->begin(),
		thrust::make_constant_iterator(beta,0))),
	thrust::make_zip_iterator(
	    thrust::make_tuple(
		((CUSPARRAY*)zin->spptr)->end(),  
		((CUSPARRAY*)xin->spptr)->end(),
		thrust::make_constant_iterator(alpha,n),	
		((CUSPARRAY*)yin->spptr)->end(),
		thrust::make_constant_iterator(beta,n))),
	VecCUDAAXPBYPZ());
    ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr); 
  } else {
    */ 
     const PetscScalar  *yy,*xx;
  PetscScalar        *zz;
  PetscInt i;
if (alpha == 1.0) {
    ierr = VecGetArrayPrivate3(xin,(PetscScalar**)&xx,yin,(PetscScalar**)&yy,zin,&zz);CHKERRQ(ierr);
    for (i=0; i<n; i++) {
      zz[i] = xx[i] + beta*yy[i] + gamma*zz[i];
    }
    ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr);
    ierr = VecRestoreArrayPrivate3(xin,(PetscScalar**)&xx,yin,(PetscScalar**)&yy,zin,&zz);CHKERRQ(ierr);
  } else if (gamma == 1.0) {
    ierr = VecGetArrayPrivate3(xin,(PetscScalar**)&xx,yin,(PetscScalar**)&yy,zin,&zz);CHKERRQ(ierr);
    for (i=0; i<n; i++) {
      zz[i] = alpha*xx[i] + beta*yy[i] + zz[i];
    }
    ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr);
    ierr = VecRestoreArrayPrivate3(xin,(PetscScalar**)&xx,yin,(PetscScalar**)&yy,zin,&zz);CHKERRQ(ierr);
  } else {
    cusp::blas::axpbypcz(*(CUSPARRAY *)(xin->spptr),*(CUSPARRAY *)(yin->spptr),*(CUSPARRAY *)(zin->spptr),*(CUSPARRAY *)(zin->spptr),alpha,beta,gamma);
    zin->valid_GPU_array = PETSC_CUDA_GPU;
    ierr = PetscLogFlops(5.0*n);CHKERRQ(ierr);    
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecPointwiseMult_SeqCUDA"
PetscErrorCode VecPointwiseMult_SeqCUDA(Vec win,Vec xin,Vec yin)
{
  PetscErrorCode ierr;
  PetscInt       n = win->map->n;

  PetscFunctionBegin;
  ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
  ierr = VecCUDACopyToGPU(yin);CHKERRQ(ierr);
  ierr = VecCUDAAllocateCheck(win);CHKERRQ(ierr);
  cusp::blas::xmy(*(CUSPARRAY *)(xin->spptr),*(CUSPARRAY *)(yin->spptr),*(CUSPARRAY *)(win->spptr));
  win->valid_GPU_array = PETSC_CUDA_GPU;
  ierr = PetscLogFlops(n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecView_SeqCUDA"
PetscErrorCode VecView_SeqCUDA(Vec xin,PetscViewer viewer)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCUDACopyFromGPU(xin);CHKERRQ(ierr);
  ierr = VecView_Seq(xin,viewer);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecNorm_SeqCUDA"
PetscErrorCode VecNorm_SeqCUDA(Vec xin,NormType type,PetscReal* z)
{
  PetscScalar    *xx;
  PetscErrorCode ierr;
  PetscInt       n = xin->map->n;
  PetscBLASInt   one = 1, bn = PetscBLASIntCast(n);

  PetscFunctionBegin;
  if (type == NORM_2 || type == NORM_FROBENIUS) {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
    *z = cusp::blas::nrm2(*(CUSPARRAY *)(xin->spptr));
    ierr = PetscLogFlops(PetscMax(2.0*n-1,0.0));CHKERRQ(ierr);
  } else if (type == NORM_INFINITY) {
    PetscInt     i;
    PetscReal    max = 0.0,tmp;

    ierr = VecGetArrayPrivate(xin,&xx);CHKERRQ(ierr);
    for (i=0; i<n; i++) {
      if ((tmp = PetscAbsScalar(*xx)) > max) max = tmp;
      /* check special case of tmp == NaN */
      if (tmp != tmp) {max = tmp; break;}
      xx++;
    }
    ierr = VecRestoreArrayPrivate(xin,&xx);CHKERRQ(ierr);
    *z   = max;
  } else if (type == NORM_1) {
    ierr = VecCUDACopyToGPU(xin);CHKERRQ(ierr);
#if defined(PETSC_USE_SCALAR_SINGLE)
    *z = hipblasSasum(bn,VecCUDACastToRawPtr(*(CUSPARRAY *)(xin->spptr)),one);
#else
    *z = hipblasDasum(bn,VecCUDACastToRawPtr(*(CUSPARRAY *)(xin->spptr)),one);
#endif
    ierr = cublasGetError();CHKERRCUDA(ierr);
    ierr = PetscLogFlops(PetscMax(n-1.0,0.0));CHKERRQ(ierr);
  } else if (type == NORM_1_AND_2) {
    ierr = VecNorm_SeqCUDA(xin,NORM_1,z);CHKERRQ(ierr);
    ierr = VecNorm_SeqCUDA(xin,NORM_2,z+1);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}


/*the following few functions should be modified to actually work with the GPU so they don't force unneccesary allocation of CPU memory */

#undef __FUNCT__  
#define __FUNCT__ "VecSetRandom_SeqCUDA"
PetscErrorCode VecSetRandom_SeqCUDA(Vec xin,PetscRandom r)
{
  PetscErrorCode ierr;
  PetscFunctionBegin;
  ierr = VecSetRandom_Seq(xin,r);CHKERRQ(ierr);
  if (xin->valid_GPU_array != PETSC_CUDA_UNALLOCATED){
    xin->valid_GPU_array = PETSC_CUDA_CPU;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecResetArray_SeqCUDA"
PetscErrorCode VecResetArray_SeqCUDA(Vec vin)
{
  PetscErrorCode ierr;
  PetscFunctionBegin;
  ierr = VecResetArray_Seq(vin);CHKERRQ(ierr);
  if (vin->valid_GPU_array != PETSC_CUDA_UNALLOCATED){
    vin->valid_GPU_array = PETSC_CUDA_CPU;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecPlaceArray_SeqCUDA"
PetscErrorCode VecPlaceArray_SeqCUDA(Vec vin,const PetscScalar *a)
{
  PetscErrorCode ierr;
  PetscFunctionBegin;
  ierr = VecPlaceArray_Seq(vin,a);CHKERRQ(ierr);
  if (vin->valid_GPU_array != PETSC_CUDA_UNALLOCATED){
    vin->valid_GPU_array = PETSC_CUDA_CPU;
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecReplaceArray_SeqCUDA"
PetscErrorCode VecReplaceArray_SeqCUDA(Vec vin,const PetscScalar *a)
{
  PetscErrorCode ierr;
  PetscFunctionBegin;
  ierr = VecReplaceArray_Seq(vin,a);CHKERRQ(ierr);
  if (vin->valid_GPU_array != PETSC_CUDA_UNALLOCATED){
    vin->valid_GPU_array = PETSC_CUDA_CPU;
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecCreateSeqCUDA"
/*@
   VecCreateSeqCUDA - Creates a standard, sequential array-style vector.

   Collective on MPI_Comm

   Input Parameter:
+  comm - the communicator, should be PETSC_COMM_SELF
-  n - the vector length 

   Output Parameter:
.  V - the vector

   Notes:
   Use VecDuplicate() or VecDuplicateVecs() to form additional vectors of the
   same type as an existing vector.

   Level: intermediate

   Concepts: vectors^creating sequential

.seealso: VecCreateMPI(), VecCreate(), VecDuplicate(), VecDuplicateVecs(), VecCreateGhost()
@*/
PetscErrorCode PETSCVEC_DLLEXPORT VecCreateSeqCUDA(MPI_Comm comm,PetscInt n,Vec *v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCreate(comm,v);CHKERRQ(ierr);
  ierr = VecSetSizes(*v,n,n);CHKERRQ(ierr);
  ierr = VecSetType(*v,VECSEQCUDA);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/*The following template functions are for VecDotNorm2_SeqCUDA.  Note that there is no complex support as currently written*/
template <typename T>
struct cudadotnormcalculate : thrust::unary_function<T,T>
{
	__host__ __device__
	T operator()(T x)
	{
		return thrust::make_tuple(thrust::get<0>(x)*thrust::get<1>(x),thrust::get<1>(x)*thrust::get<1>(x));
	}
};

template <typename T>
struct cudadotnormreduce : thrust::binary_function<T,T,T>
{
	__host__ __device__
	T operator()(T x,T y)
	{
		return thrust::make_tuple(thrust::get<0>(x)+thrust::get<0>(y),thrust::get<1>(x)+thrust::get<1>(y));
	}
};
	
#undef __FUNCT__
#define __FUNCT__ "VecDotNorm2_SeqCUDA"
PetscErrorCode VecDotNorm2_SeqCUDA(Vec s, Vec t, PetscScalar *dp, PetscScalar *nm)
{
  PetscErrorCode                         ierr;
  PetscScalar                            zero = 0.0,n=s->map->n;
  thrust::tuple<PetscScalar,PetscScalar> result;

  PetscFunctionBegin;
  ierr = VecCUDACopyToGPU(s);CHKERRQ(ierr);
  ierr = VecCUDACopyToGPU(t);CHKERRQ(ierr);
  result = thrust::transform_reduce(thrust::make_zip_iterator(thrust::make_tuple(((CUSPARRAY *)s->spptr)->begin(),((CUSPARRAY *)t->spptr)->begin())),
				    thrust::make_zip_iterator(thrust::make_tuple(((CUSPARRAY *)s->spptr)->end(),((CUSPARRAY *)t->spptr)->end())),
				    cudadotnormcalculate<thrust::tuple<PetscScalar,PetscScalar> >(),
				    thrust::make_tuple(zero,zero), /*init */
				    cudadotnormreduce<thrust::tuple<PetscScalar, PetscScalar> >()); /* binary function */
  *dp = thrust::get<0>(result);
  *nm = thrust::get<1>(result);
  ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecDuplicate_SeqCUDA"
PetscErrorCode VecDuplicate_SeqCUDA(Vec win,Vec *V)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecCreateSeqCUDA(((PetscObject)win)->comm,win->map->n,V);CHKERRQ(ierr);
  if (win->mapping) {
    ierr = PetscObjectReference((PetscObject)win->mapping);CHKERRQ(ierr);
    (*V)->mapping = win->mapping;
  }
  if (win->bmapping) {
    ierr = PetscObjectReference((PetscObject)win->bmapping);CHKERRQ(ierr);
    (*V)->bmapping = win->bmapping;
  }
  (*V)->map->bs = win->map->bs;
  ierr = PetscOListDuplicate(((PetscObject)win)->olist,&((PetscObject)(*V))->olist);CHKERRQ(ierr);
  ierr = PetscFListDuplicate(((PetscObject)win)->qlist,&((PetscObject)(*V))->qlist);CHKERRQ(ierr);

  (*V)->stash.ignorenegidx = win->stash.ignorenegidx;
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecDestroy_SeqCUDA"
PetscErrorCode VecDestroy_SeqCUDA(Vec v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  delete (CUSPARRAY *)(v->spptr);
  ierr = VecDestroy_Seq(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

EXTERN_C_BEGIN
#undef __FUNCT__  
#define __FUNCT__ "VecCreate_SeqCUDA"
PetscErrorCode PETSCVEC_DLLEXPORT VecCreate_SeqCUDA(Vec V)
{
  PetscErrorCode ierr;
  PetscMPIInt    size;
 
  PetscFunctionBegin;
  ierr = MPI_Comm_size(((PetscObject)V)->comm,&size);CHKERRQ(ierr);
  if  (size > 1) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Cannot create VECSEQCUDA on more than one process");
  ierr = VecCreate_Seq_Private(V,0);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)V,VECSEQCUDA);CHKERRQ(ierr);
  V->ops->duplicate       = VecDuplicate_SeqCUDA;
  V->ops->dot             = VecDot_SeqCUDA;
  V->ops->norm            = VecNorm_SeqCUDA;
  V->ops->tdot            = VecTDot_SeqCUDA;
  V->ops->scale           = VecScale_SeqCUDA;
  V->ops->copy            = VecCopy_SeqCUDA;
  V->ops->set             = VecSet_SeqCUDA;
  V->ops->swap            = VecSwap_SeqCUDA;
  V->ops->axpy            = VecAXPY_SeqCUDA;
  V->ops->axpby           = VecAXPBY_SeqCUDA;
  V->ops->axpbypcz        = VecAXPBYPCZ_SeqCUDA;
  V->ops->pointwisemult   = VecPointwiseMult_SeqCUDA;
  V->ops->pointwisedivide = VecPointwiseDivide_SeqCUDA;
  V->ops->setrandom       = VecSetRandom_SeqCUDA;
  V->ops->view            = VecView_SeqCUDA;
  V->ops->placearray      = VecPlaceArray_SeqCUDA;
  V->ops->replacearray    = VecReplaceArray_SeqCUDA;
  V->ops->dot_local       = VecDot_SeqCUDA;
  V->ops->tdot_local      = VecTDot_SeqCUDA;
  V->ops->norm_local      = VecNorm_SeqCUDA;
  V->ops->resetarray      = VecResetArray_SeqCUDA;
  V->ops->destroy         = VecDestroy_SeqCUDA;
  V->ops->maxpy           = VecMAXPY_SeqCUDA;
  V->ops->mdot            = VecMDot_SeqCUDA;
  V->ops->aypx            = VecAYPX_SeqCUDA;
  V->ops->waxpy           = VecWAXPY_SeqCUDA;
  /*V->ops->dotnorm2        = VecDotNorm2_SeqCUDA;*/
  V->valid_GPU_array      = PETSC_CUDA_UNALLOCATED;
  PetscFunctionReturn(0);
}
EXTERN_C_END
